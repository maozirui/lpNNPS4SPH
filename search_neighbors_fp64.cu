
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 1000
#define R_CUT 0.2
#define max_neighbor 1000

__global__ void search_neighbors(double *x, double *y, double *z, int *neighbors, int *num_neighbors) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int count = 0;
    for (int j = 0; j < N; j++) {
        if (i != j) {
            double dx = x[i] - x[j];
            double dy = y[i] - y[j];
            double dz = z[i] - z[j];
            double r = sqrt(dx*dx + dy*dy + dz*dz);
            if (r <= R_CUT) {
                neighbors[i*max_neighbor + count] = j;
                count++;
            }
        }
    }
    num_neighbors[i] = count;
}

int main() {
    double x[N], y[N], z[N];
    int neighbors[N*max_neighbor], num_neighbors[N];
    FILE *fp;

    fp = fopen("coordinates_1000.dat", "r");

    for (int i = 0; i < N; i++) {
        fscanf(fp, "%lf %lf %lf", &x[i], &y[i], &z[i]);
    }

    fclose(fp);

    // for (int i = 0; i < N; i++) {
    //     x[i] = rand() / (double)RAND_MAX;
    //     y[i] = rand() / (double)RAND_MAX;
    //     z[i] = rand() / (double)RAND_MAX;
    // }

    double *d_x, *d_y, *d_z;
    int *d_neighbors, *d_num_neighbors;

    hipMalloc((void **)&d_x, N*sizeof(double));
    hipMalloc((void **)&d_y, N*sizeof(double));
    hipMalloc((void **)&d_z, N*sizeof(double));
    hipMalloc((void **)&d_neighbors, N*max_neighbor*sizeof(int));
    hipMalloc((void **)&d_num_neighbors, N*sizeof(int));

    hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, N*sizeof(double), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int i = 0; i < 5000; i++) {
        search_neighbors<<<grid_size, block_size>>>(d_x, d_y, d_z, d_neighbors, d_num_neighbors);
    }

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float elapsed_time_ms;
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    

    hipMemcpy(neighbors, d_neighbors, N*max_neighbor*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(num_neighbors, d_num_neighbors, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = N-1; i < N; i++) {
        printf("Particle %d neighbors: ", i);
        for (int j = 0; j < num_neighbors[i]; j++) {
            printf("%d ", neighbors[i*max_neighbor + j]);
        }
        printf("\n");
    }

    printf("Elapsed time: %.3f ms\n", elapsed_time_ms);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_neighbors);
    hipFree(d_num_neighbors);

    return 0;
}
